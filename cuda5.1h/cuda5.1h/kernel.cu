#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "cmath"
#include <stdio.h>

#define N 5 

__global__ void ProdD(double* a, double* b, double* c)
{
	int i = threadIdx.x; 
	if (i > N - 1) return; 	
	c[i] = __dmul_rn(a[i], b[i]);
}

__global__ void ProdF(float* a, float* b, float* c)
{
	int i = threadIdx.x; 
	if (i > N - 1) return; 
	c[i] = __fmul_rn(a[i], b[i]);
}

int main()
{
	hipEvent_t start_f, stop_f, start_d, stop_d;
	hipEventCreate(&start_f);
	hipEventCreate(&stop_f);
	hipEventCreate(&start_d);
	hipEventCreate(&stop_d);

	float a_f[N] = { 1,2,3,4,5 }, b_f[N]= { 1,2,3,4,5 }, c_f[N];
	double a_d[N]= { 1,2,3,4,5 }, b_d[N]= { 1,2,3,4,5 }, c_d[N];

	float* dev_a_f, * dev_b_f, * dev_c_f;
	double* dev_a_d, * dev_b_d, * dev_c_d;



	hipMalloc((void**)&dev_a_f, N * sizeof(float));
	hipMalloc((void**)&dev_b_f, N * sizeof(float));
	hipMalloc((void**)&dev_c_f, N * sizeof(float));
	hipMalloc((void**)&dev_a_d, N * sizeof(double));
	hipMalloc((void**)&dev_b_d, N * sizeof(double));
	hipMalloc((void**)&dev_c_d, N * sizeof(double));

	hipMemcpy(dev_a_f, a_f, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b_f, b_f, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_a_d, a_d, N * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_b_d, b_d, N * sizeof(double), hipMemcpyHostToDevice);

	hipEventRecord(start_f, 0); 
	ProdF << <1, N >> > (dev_a_f, dev_b_f, dev_c_f);
	hipEventRecord(stop_f, 0); 
	hipEventSynchronize(stop_f);

	float kernelTime_f;
	hipEventElapsedTime(&kernelTime_f, start_f, stop_f);
	printf("Float kernel time = %f ms\n", kernelTime_f);

	hipEventRecord(start_d, 0); 
	ProdD << <1, N >> > (dev_a_d, dev_b_d, dev_c_d);
	hipEventRecord(stop_d, 0); 
	hipEventSynchronize(stop_d);

	float kernelTime_d;
	hipEventElapsedTime(&kernelTime_d, start_d, stop_d);
	printf("Double kernel time = %f ms\n", kernelTime_d);

	hipMemcpy(c_f, dev_c_f, N * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(c_d, dev_c_d, N * sizeof(double), hipMemcpyDeviceToHost);

	float prod_f = 0;
	double prod_d = 0;

	for (int i = 0; i < N; i++)
	{
		prod_f += c_f[i];
		prod_d += c_d[i];
	}

	printf("prod_f = %f\nprod_d = %f\n", prod_f, prod_d);

	hipFree(dev_a_f);
	hipFree(dev_b_f);
	hipFree(dev_c_f);
	hipFree(dev_a_d);
	hipFree(dev_b_d);
	hipFree(dev_c_d);
	return 0;
}