#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <stdio.h>
#define N 1000 

__global__ void ZFunction(float* a, float* b)
{
    int i = threadIdx.x; 
    a[i] = 1.f / powf(float(i + 1), *b);
}


int main()
{
    float b = 2; //степень
    float a[N]; //массив членов ряда
    float* dev_b = 0;
    float* dev_a = 0;
    float sum = 0; //частная сумма ряда

    hipMalloc((void**)&dev_a, N * sizeof(float));
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) printf("%s ",
        hipGetErrorString(err));
    hipMalloc((void**)&dev_b, sizeof(float));
    if (err != hipSuccess) printf("%s ",
        hipGetErrorString(err));
    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("%s ",
        hipGetErrorString(err));
    hipMemcpy(dev_b, &b, sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("%s ",
        hipGetErrorString(err));
    ZFunction << <1, N >> > (dev_a, dev_b);

    hipMemcpy(a, dev_a, N * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) printf("%s ",
        hipGetErrorString(err));
    for (int i = 0; i < N; ++i)
    {
        sum += a[i];
    }
    printf("%f\n", sum);

    hipFree(dev_a);
    hipFree(dev_b);
    return 0;
}