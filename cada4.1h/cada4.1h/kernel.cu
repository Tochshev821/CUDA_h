#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <stdio.h>

#define N 1000


__device__ bool IsApropriate(double* x, double* y)
{
    bool result = (*x) * (*x) + (*y) * (*y) <= 1;
    return result;
}

__global__ void CalculatePI(int * dev_a)
{
    double x = (double)blockIdx.x / N;
    double y = (double)threadIdx.x / N;
    IsApropriate(&x, &y) ? atomicAdd(dev_a, 1) : 0;
}

int main()
{
    int di = N * N;
    int a = 0;
    int* dev_a;

    hipMalloc((void**)&dev_a, sizeof(int));
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) printf("%s ",
        hipGetErrorString(err));
    hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("%s ",
        hipGetErrorString(err));
    CalculatePI << < N,  N>> > (dev_a);

    hipMemcpy(&a, dev_a, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) printf("%s ",
        hipGetErrorString(err));
    printf("pi = %f\n", (double)a * 4 / di);

    hipFree(dev_a);
    return 0;
}