#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <stdio.h>
#define N 10


__global__ void matrixAdd(const int* A, const
    int* B, int* C)
{
    int i = blockIdx.x;
    int j = threadIdx.x;
    C[i * N + j] = A[i * N + j] + B[i * N + j];
}

int main()
{
    int a[N][N];
    int b[N][N];
    int c[N][N];
    int* ca;
    int* cb;
    int* cc;
    for (int i = 0; i < N * N; ++i) {
        *(*a + i) = 3;
    }

    for (int i = 0; i < N * N; ++i) {
        *(*b + i) = 3;
    }
    hipMalloc((void**)&ca, N * N * sizeof(int));
    hipMalloc((void**)&cb, N * N * sizeof(int));
    hipMalloc((void**)&cc, N * N * sizeof(int));
    hipMemcpy(ca, &a, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(cb, &b, N * N * sizeof(int), hipMemcpyHostToDevice);

    matrixAdd << <N, N >> > (ca, cb, cc);
    hipMemcpy(&c, cc, N * N * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%d ", c[i][j]);
        }
        printf("\n");
    }

    hipFree(ca);
    hipFree(cb);
    hipFree(cc);
    return 0;
}